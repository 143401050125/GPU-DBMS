#include "yacc.cu"
#include "lex.cu"

int main()
{
	dbpath = "";
	tmp_table = 0; //assuming tmp table path is dbpath + "/tmp/table_" + tmp_table;
	print_tables = true;
	tmp_table_limit = 100;
	do 
	{
		setjmp(env_buffer);
		char *line = NULL;
		size_t len = 0;
		std::cout<<"Enter Query : ";
		getline(&line, &len, stdin);
		YY_BUFFER_STATE buffer = yy_scan_string(line);
		yy_switch_to_buffer(buffer);
		hipEventCreate(&start); 
		hipEventCreate(&stop);
		hipEventRecord(start);
		yyparse();
		yy_delete_buffer(buffer);
	} while (!feof(stdin));
	
	return 0;
}