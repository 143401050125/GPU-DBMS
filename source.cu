#include "header.h"

hipEvent_t start, stop;
float et;
jmp_buf env_buffer;
std::string dbpath;
int tmp_table;
bool print_tables;
int tmp_table_limit;
std::set<table *> all_table;

column::column()
{
  type = -1;
  tname = "";
}

table::table(std::string nn)
{
  all_table.insert(this);
  name = nn;
  original_name = nn;
  flag = false;
  umap.clear();
  row_count = -1;
  if(nn != "")
  {
    std::ifstream f;
    std::string table_name = dbpath + original_name + ".txt";
    f.open(table_name);
    if(!f.is_open())
    {
      yyerror(name + " : No such table2 ");
    }
    std::string cur_word, meta;
    int num_rows;
    int num_cols;
    f >> num_rows;
    f >> num_cols;
    row_count = num_rows;
    getline(f,meta);
    f.close();
    std::stringstream s(meta);
    while(num_cols--)
    {
        s >> cur_word;
        columnNames.insert(cur_word);
        s >> cur_word;
        s >> cur_word;
    }	
  }
}
std::string table::get_column_name(std::string colname)
{
  size_t found = colname.find_first_of(".");
  std::string tname = colname.substr(0,found);
  std::string cname = colname.substr(found+1);
  if(found != std::string::npos && tname == name)
    colname = cname;
  return colname;
}
column& table::get_column(std::string col)
{
  col = this->get_column_name(col);
  if(umap.find(col) != umap.end())
    return umap[col];
      
  if(original_name == "")
    yyerror(col + " : Column not found.");
  
  std::ifstream f;
  std::string table_name = dbpath + original_name + ".txt";
  f.open(table_name);
  
  if(!f.is_open())
    yyerror(name + " : No such table3 ");
  
  std::string cur_word, col_type;
  int get_num, num_rows, num_cols, offset_of_col, is_present;
  bool flag = false;
  f >> num_rows;
  f >> num_cols;
  while(num_cols--)
  {
    f >> cur_word;
    if(cur_word == col)
      {
        f >> col_type;
        f >> offset_of_col;
        flag = true;
        break;
      }
    else
    {
        f >> cur_word;
        f >> get_num;
    }
  }
  if(flag)
  {
      column &newCol = umap[col];
      newCol.tname = name;
      f.seekg(offset_of_col,std::ios::beg);
      f >> is_present;
      if(is_present)
        f >> newCol.tname;
      int row_iter = 0;
      if(col_type == "int")
      {
          newCol.type = 0;
          thrust::host_vector<int> h(num_rows);
          while(row_iter < num_rows)
          {
            f >> h[row_iter++];
          }
          newCol.i = h;
      }
      else
      {
          newCol.type = 1;
          thrust::host_vector<float> h(num_rows);
          while(row_iter < num_rows)
          {
            f >> h[row_iter++];
          }
          newCol.f = h;
      }
  }
  else
  {
    {
      std::cout<<"column of " + original_name <<std::endl;
      for(auto c:columnNames)
        std::cout<<c<<" ";
      std::cout<<std::endl;
    }
    yyerror(col + " : No such column found in database -- '" + name + "' ---- " + original_name);
  }
  f.close();
  
  if(key.size() != 0)
  {
    column &col_ = umap[col];
    if(col_.type)
    {
      thrust::device_vector<float>::iterator it_end;
      it_end = thrust::remove_if(col_.f.begin(),col_.f.end(),key.begin(),thrust::logical_not<bool>());
      col_.f.resize(it_end - col_.f.begin());
    }else
    {
      thrust::device_vector<int>::iterator it_end;
      it_end = thrust::remove_if(col_.i.begin(),col_.i.end(),key.begin(),thrust::logical_not<bool>());
      col_.i.resize(it_end - col_.i.begin());
    }
  }
  return umap[col];
}

void table::set_column(std::string colname,column &col)
{
  colname = this->get_column_name(colname);
  umap[colname] = col;
  if(col.type)
    row_count = col.f.size();
  else
    row_count = col.i.size();
  columnNames.insert(colname);
}

void table::print(std::vector<std::string> &col_order,int row_limit)
{
  std::cout<<std::endl;
  class col{
    public:
      int type;
      thrust::host_vector<int> i;
      thrust::host_vector<float> f;
  };
  std::unordered_map<std::string,col>::iterator it;
  std::unordered_map<std::string,col> umap;
 
  if(col_order.size() == 0)
  {
    for(auto cname:columnNames)
      col_order.push_back(cname);
  }
 
  for(auto &p: this->umap)
  {
    umap[p.first].type = p.second.type;
    if(p.second.type)
      umap[p.first].f = p.second.f;
    else
      umap[p.first].i = p.second.i;
  }
  
  int tot_row = row_count;
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&et, start, stop);

  if(tot_row == 0)
  {
    std::cout<<"Empty set in "<<et/1000<<" seconds."<<std::endl;
    return;
  }
  
  TextTable t( '-', '|', '+' );
  for(auto cname : col_order)
    t.add(cname);
  t.endOfRow();
  int row_max = tot_row;
  if(row_limit != -1 && row_limit < row_max)
      row_max = row_limit;
  if(print_tables)
  {
    for(int cur_row = 0; cur_row < row_max; cur_row++)
    {	
      for(auto cname:col_order)
      {
        col &c = umap[this->get_column_name(cname)];
        if(c.type)
          t.add( (c.f[cur_row] != FLOAT_FLAG) ? std::to_string(c.f[cur_row]) : "NULL");
        else
          t.add( (c.i[cur_row] != INT_FLAG) ? std::to_string(c.i[cur_row]) : "NULL");
      }
      t.endOfRow();
    }
    t.setAlignment( 2, TextTable::Alignment::LEFT );
    std::cout << t;
  } 
  std::cout<<tot_row<< " rows in "<<et/1000<<" seconds."<<std::endl;
}

int table::size()
{
  return umap.size();
}

column& table::get_first_column()
{
  if(umap.size() == 0)
    yyerror("Table is empty.");
  return umap.begin()->second;
}

std::string table::get_first_column_name()
{
  if(umap.size() == 0)
    yyerror("Table is empty.");
  return umap.begin()->first;
}
column& table::new_column(std::string cname)
{
  cname = this->get_column_name(cname);
  if(columnNames.find(cname) != columnNames.end() || umap.find(cname) != umap.end())
    yyerror(cname + " : Column already exist");
  columnNames.insert(cname);
  return umap[cname];
}

void table::erase_column(std::string cname)
{
  cname = this->get_column_name(cname);
  if(columnNames.find(cname) == columnNames.end() && umap.find(cname) == umap.end())
    yyerror(cname + " : No such Column to erase");
  columnNames.erase(cname);
  umap.erase(cname);
}

void table::print_column()
{
  std::cout<<"Column of '"<<name<<"' :"<<std::endl;
  for(auto p:umap)
    std::cout<<p.first<<" ";
  std::cout<<std::endl;
}

void table::updatekey(table &t1)
{
  if(t1.size() != 1)
    yyerror("Invalid Opeartion : key is not present");
  column &col = t1.get_first_column();
  if(col.type)
  {
    key.resize(col.f.size());
    thrust::transform(col.f.begin(),col.f.end(),key.begin(),[=] __device__ __host__  (float &f) { return (f==0) ? false : true;});
  }
  else
  {
    key.resize(col.i.size());
    thrust::transform(col.i.begin(),col.i.end(),key.begin(),[=] __device__ __host__ (int &i) { return (i==0) ? false : true;});
  }
  
  row_count = thrust::count_if(key.begin(),key.end(),thrust::identity<bool>());
  //applying key on loaded columns 
  for(auto &p:umap)
  {
    column &col = p.second;
    if(col.type)
    {
      thrust::device_vector<float>::iterator it_end;
      it_end = thrust::remove_if(col.f.begin(),col.f.end(),key.begin(),thrust::logical_not<bool>());
      col.f.resize(it_end - col.f.begin());
    }
    else
    {
      thrust::device_vector<int>::iterator it_end;
      it_end = thrust::remove_if(col.i.begin(),col.i.end(),key.begin(),thrust::logical_not<bool>());
      col.i.resize(it_end - col.i.begin());
    }
  }
}

void table::move_column(std::string cname1,std::string cname2)
{
  cname1 = this->get_column_name(cname1);
  cname2 = this->get_column_name(cname2);
  umap[cname2] = std::move(umap[cname1]);
  umap.erase(cname1);
  /*auto nodeHandler = umap.extract(cname1);
  nodeHandler.key() = cname2;
  umap.insert(std::move(nodeHandler));*/
  columnNames.erase(cname1);
  columnNames.insert(cname2);
}

void table::copy_column(std::string cname,column &col)
{
  cname = this->get_column_name(cname);
  umap[cname] = std::move(col);
  columnNames.insert(cname);
}

void table::write_metadata(std::string tname,int nrows, int ncols)
{
  if(tname != "")
    original_name = tname;
  //write information related to table 
  //-------useless-----write meta data using use column names from s1 and s2;
  std::ofstream f;
  std::string table_name_cur = dbpath + tname + ".txt";
  f.open(table_name_cur);
  if(!f.is_open())
  {
    yyerror(name + " : Unable to write temporaray table");
  }
  std::string to_writ, cur_wrd;
  to_writ = std::to_string(nrows) + " " + std::to_string(ncols);
  rjust(to_writ,999);
  to_writ += "\n";
  f << to_writ;
  f.close();
}

void table::write_column(std::string cname)
{
  //write metadata and values 
  //of this->get_column(cname) in to file
  column &tcol = this->get_column(cname);
  std::ifstream f;
  std::string table_name = "./" + original_name + ".txt";
  f.open(table_name);
  if(!f.is_open())
  {
    yyerror(name + " : No such table to write column ");
  }
  std::string to_add, meta;
  int nrows,ncols;
  f >> nrows;
  f >> ncols;
  getline(f,meta);
  f.close();
  
  to_add = " "+cname;
  if(tcol.type)
    to_add += " float ";
  else
    to_add += " int ";
  std::string fin_col = "", ele_str;
  fin_col = "1 ";
  rjust(fin_col,12);
  ele_str = tcol.tname;
  fin_col += (ele_str + " ");

  if(tcol.type)
  {
      thrust::host_vector<float> f = tcol.f;
      tcol.f.clear();
      for(int c_it = 0;c_it < nrows;c_it++)
      {
        ele_str = std::to_string(f[c_it]);
        rjust(ele_str,12);
        if(c_it == (nrows-1))
          fin_col += ele_str + "\n";
        else
          fin_col += ele_str + " ";
      }
  }
  else
  {
      thrust::host_vector<int> i = tcol.i;
      tcol.i.clear();
      for(int c_it = 0;c_it < nrows;c_it++)
      {
        ele_str = std::to_string(i[c_it]);
        rjust(ele_str,12);
        if(c_it == (nrows-1))
          fin_col += ele_str + "\n";
        else
          fin_col += ele_str + " ";
      }
  }
  std::ofstream fot;
  fot.open(table_name,std::ios::in | std::ios::out);
  fot.seekp(0,std::ios::end);
  int last = fot.tellp();
  to_add += std::to_string(last);
  to_add += "\n"; 
  std::string fin_meta = std::to_string(nrows) + " " + std::to_string(ncols+1) + " " + meta + to_add;
  rjust(fin_meta,1000);
  fot << fin_col;
  fot.seekp(0,std::ios::beg);
  fot << fin_meta;
  fot.close();
  umap.erase(cname);	//dont erase cname from this->columnNames 
}

void table::write(std::string tname = "")
{
  this->write_metadata(tname);
  for(auto cname:this->columnNames)
    this->write_column(cname);
}

void table::clear()
{
  columnNames.clear();
  for(auto &p:umap)
    p.second.i.clear(),p.second.f.clear();
  umap.clear();
}