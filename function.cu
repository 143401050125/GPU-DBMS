#include "hip/hip_runtime.h"
#include "header.h"

node * makenode(std::string id,std::string name, node *c1 , node *c2 , node *c3 , node *c4 , node *c5 , node *c6 , node *c7 , node *c8 , node *c9 , node *c10)
{
  node * nn = new node;
  nn->name = new std::string;
  nn->id = new std::string;
  *(nn->id) = id;
  *(nn->name) = name;
  
  nn->child[0] = c1;
  nn->child[1] = c2;
  nn->child[2] = c3;
  nn->child[3] = c4;
  nn->child[4] = c5;
  nn->child[5] = c6;
  nn->child[6] = c7;
  nn->child[7] = c8;
  nn->child[8] = c9;
  nn->child[9] = c10;
  
  nn->size = 0;
  for(int i=0;i<10;i++)
    if(nn->child[i])
      nn->size++;
  
  return nn;
  if(c1)
    nn->child[0] = c1,nn->size++;
  if(c2)
    nn->child[1] = c2,nn->size++;
  if(c3)
    nn->child[2] = c3,nn->size++;
  if(c4)
    nn->child[3] = c4,nn->size++;
  if(c5)
    nn->child[4] = c5,nn->size++;
  if(c6)
    nn->child[5] = c6,nn->size++;
  if(c7)
    nn->child[6] = c7,nn->size++;
  if(c8)
    nn->child[7] = c8,nn->size++;
  if(c9)
    nn->child[8] = c9,nn->size++;
  if(c10)
    nn->child[9] = c10,nn->size++;

  return nn;
}

void yyerror(std::string s) 
{
	std::cout<<std::endl<<s<<std::endl<<std::endl;
	longjmp(env_buffer, 1);
}

void rjust(std::string &orig, int siz)
{
		int cur_len = orig.length();
		siz -= cur_len;
		orig = std::string(siz,' ') + orig;
}

void binary_op(table & t1, table & t2,const std::string &op)
{
	if(!(t1.size() == 1 && t2.size() == 1))
		yyerror("Invalid operation : operand is not a column (binary operation)");
	
	column &col1 = t1.get_first_column();
	column &col2 = t2.get_first_column();
	
	if(col1.type != col2.type)
	{
		if(col1.type)
		{
			thrust::device_vector<float> &key1 = col2.f;
			to_float funt;
			key1.resize(col2.i.size());
			thrust::transform(col2.i.begin(),col2.i.end(),key1.begin(),funt);
			col2.i.clear();
			col2.type = 1;
		}else
		{
			thrust::device_vector<float> &key1 = col1.f;
			to_float funt;
			key1.resize(col1.i.size());
			thrust::transform(col1.i.begin(),col1.i.end(),key1.begin(),funt);
			col1.i.clear();
			col1.type = 1;
		}
	}
	
	if(col1.type)
	{
		if(col1.f.size() != col2.f.size())
		{
			if(col1.f.size() == 1 && col2.f.size() > 0)
			{
				col1.f.resize(col2.f.size(),col1.f[0]);
				t1.row_count = t2.row_count;
			}else if(col2.f.size() == 1 && col1.f.size() > 0)
			{
				col2.f.resize(col1.f.size(),col2.f[0]);
				t2.row_count = t1.row_count;
			}else
			{
				yyerror("Invalid operation : operand(columns) size is not equal");
				return;
			}
		}
	}else
	{
		if(col1.i.size() != col2.i.size())
		{
			if(col1.i.size() == 1 && col2.i.size() > 0)
			{
				col1.i.resize(col2.i.size(),col1.i[0]);
				t1.row_count = t2.row_count;
			}else if(col2.i.size() == 1 && col1.i.size() > 0)
			{
				col2.i.resize(col1.i.size(),col2.i[0]);
				t2.row_count = t1.row_count;
			}else
			{
				yyerror("Invalid operation : operand(columns) size is not equal");
				return;
			}
		}
	}
	#undef TRANSFORM
	#define TRANSFORM(op) thrust::transform(key1.begin(),key1.end(),key2.begin(),key1.begin(),op)
	
	assert(t1.row_count == t2.row_count);
	thrust::device_vector<bool> is_null(t1.row_count);	

	if(col1.type)
	{
		thrust::device_vector<float> &key1 = col1.f;
		thrust::device_vector<float> &key2 = col2.f;

		thrust::transform(key1.begin(),key1.end(),key2.begin(),is_null.begin(),[=] __device__ __host__  (float &f1,float &f2) { return (f1==FLOAT_FLAG || f2==FLOAT_FLAG) ? false : true;});
		
		if (op == "NEQ")
			TRANSFORM(thrust::not_equal_to<float>());
		else if (op == ">")
			TRANSFORM(thrust::greater<float>());
		else if (op == "<")
			TRANSFORM(thrust::less<float>());
		else if (op == "GEQ")
			TRANSFORM(thrust::greater_equal<float>());
		else if (op == "LEQ")
			TRANSFORM(thrust::less_equal<float>());
		else if (op == "EQ")
			TRANSFORM(thrust::equal_to<float>());
		else if (op == "OR")
			TRANSFORM(thrust::logical_or<float>());
		else if (op == "AND")
			TRANSFORM(thrust::logical_and<float>());
		else if (op == "+")
			TRANSFORM(thrust::plus<float>());
		else if (op == "-")
			TRANSFORM(thrust::minus<float>());
		else if (op == "*")
			TRANSFORM(thrust::multiplies<float>());
		else if (op == "/")
			TRANSFORM(thrust::divides<float>());
		else if (op == "%")
		{
			fmodulus f;
			TRANSFORM(f);
		}
		else 
			yyerror("Undefined Binary Operation");
		if(op == "+" || op == "-" || op == "*" || op == "/" || op == "%")
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,float &f) { return (b==false) ? FLOAT_FLAG : f;});
		else
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,float &f) { return (b==false) ? 0 : f;});
	}else
	{
		thrust::device_vector<int> &key1 = col1.i;
		thrust::device_vector<int> &key2 = col2.i;

		thrust::transform(key1.begin(),key1.end(),key2.begin(),is_null.begin(),[=] __device__ __host__  (int &i1,int &i2) { return (i1==INT_FLAG || i2==INT_FLAG) ? false : true;});
		
		if (op == "NEQ")
			TRANSFORM(thrust::not_equal_to<int>());
		else if (op == ">")
			TRANSFORM(thrust::greater<int>());
		else if (op == "<")
			TRANSFORM(thrust::less<int>());
		else if (op == "GEQ")
			TRANSFORM(thrust::greater_equal<int>());
		else if (op == "LEQ")
			TRANSFORM(thrust::less_equal<int>());
		else if (op == "EQ")
			TRANSFORM(thrust::equal_to<int>());
		else if (op == "OR")
			TRANSFORM(thrust::logical_or<int>());
		else if (op == "AND")
			TRANSFORM(thrust::logical_and<int>());
		else if (op == "+")
			TRANSFORM(thrust::plus<int>());
		else if (op == "-")
			TRANSFORM(thrust::minus<int>());
		else if (op == "*")
			TRANSFORM(thrust::multiplies<int>());
		else if (op == "/")
			TRANSFORM(thrust::divides<int>());
		else if (op == "%")
			TRANSFORM(thrust::modulus<int>());
		else 
			yyerror("Undefined Binary Operation");
		
		if(op == "+" || op == "-" || op == "*" || op == "/" || op == "%")
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,int &i) { return (b==false) ? INT_FLAG : i;});
		else
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,int &i) { return (b==false) ? 0 : i;});
	}
	
	is_null.clear();

}


void unary_op(table & t1, std::string &op)
{
	if(t1.size() != 1)
	{
		yyerror("Invalid operation : operand is not a column (unary operation)");
		return;
	}
	column &col1 = t1.get_first_column();
	thrust::device_vector<bool> is_null(t1.row_count,false);	
	
	if(col1.type)
	{
		thrust::device_vector<float> &key1 = col1.f;
		thrust::transform(key1.begin(),key1.end(),is_null.begin(),[=] __device__ __host__  (float &f) { return (f==FLOAT_FLAG) ? false : true;});
		if (op == "!")
			thrust::transform(key1.begin(),key1.end(),key1.begin(),thrust::logical_not<float>());
		else if (op == "-")
			thrust::transform(key1.begin(),key1.end(),key1.begin(),thrust::negate<float>());
		else if(op != "+")
			yyerror("Undefined Unary Operation");

		if(op == "!")
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,float &f) { return (b==false) ? 0 : f;});
		else
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,float &f) { return (b==false) ? FLOAT_FLAG : f;});
	}else
	{
		thrust::device_vector<int> &key1 = col1.i;
		thrust::transform(key1.begin(),key1.end(),is_null.begin(),[=] __device__ __host__  (int &i) { return (i==INT_FLAG) ? false : true;});
		if (op == "!")
			thrust::transform(key1.begin(),key1.end(),key1.begin(),thrust::logical_not<int>());
		else if (op == "-")
			thrust::transform(key1.begin(),key1.end(),key1.begin(),thrust::negate<int>());
		else if(op != "+")
			yyerror("Undefined Unary Operation");
		
		if(op == "!")
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,int &i) { return (b==false) ? 0 : i;});
		else
			thrust::transform(is_null.begin(),is_null.end(),key1.begin(),key1.begin(),[=] __device__ __host__  (bool &b,int &i) { return (b==false) ? INT_FLAG : i;});
	}
	is_null.clear();
}


void apply_result(table &t1, table &t2)
{
	int row_count = t1.row_count;
	assert(t1.row_count == t2.row_count);
	if(t2.size() != 1)
		yyerror("Invalid Opeartion : key is not present");
	
	bool flag1 = false,flag2 = false;
	column &key = t2.get_first_column();
	if(key.type)
	{
		if(key.f.size() == 1)
		{
			flag1 = true;
			flag2 = (key.f[0] != 0.0);
		}else if(key.f.size() != row_count)
		{
			yyerror("key size does not match column size");
			return;
		}
	}else
	{
		if(key.i.size() == 1)
		{
			flag1 = true;
			flag2 = (key.i[0] != 0);
		}else if(key.i.size() != row_count)
		{
			yyerror("key size does not match column size");
			return;
		}
	}

	for(auto cname:t1.columnNames)
	{
		column &col = t1.get_column(cname);
		if(col.type)
		{
			if(col.f.size() == 1)
			{
				col.f.resize(row_count,col.f[0]);
			}else if(col.f.size() != row_count)
			{
				yyerror("Column " + cname + " has elements not equal to rowcount of table.");
				return;
			}

			if(flag1)
			{
				if(flag2 == false)
					col.f.clear();
			}else
			{
				thrust::device_vector<float>::iterator it_end;
				if(key.type)
					it_end = thrust::remove_if(col.f.begin(),col.f.end(),key.f.begin(),thrust::logical_not<float>());
				else
					it_end = thrust::remove_if(col.f.begin(),col.f.end(),key.i.begin(),thrust::logical_not<int>());
				int newCount = it_end - col.f.begin();
				col.f.resize(newCount);
				t1.row_count = newCount;
			}
		}else
		{
			if(col.i.size() == 1)
			{
				col.i.resize(row_count,col.i[0]);
			}else if(col.i.size() != row_count)
			{
				yyerror("column " + cname + " has elements not equal to rowcount of table");
				return;
			}

			if(flag1)
			{
				if(flag2 == false)
					col.i.clear();
			}else
			{
				thrust::device_vector<int>::iterator it_end;
				if(key.type)
					it_end = thrust::remove_if(col.i.begin(),col.i.end(),key.f.begin(),thrust::logical_not<float>());
				else
					it_end = thrust::remove_if(col.i.begin(),col.i.end(),key.i.begin(),thrust::logical_not<int>());
				int newCount = it_end - col.i.begin();
				col.i.resize(newCount);
				t1.row_count = newCount;
			}
		}
	}
}

table &cross_prod(table &t1,table &t2)
{
	table &t = *(new table);
	t.row_count = t1.row_count * t2.row_count;
	bool write = false;
	if(t.row_count > tmp_table_limit)
 	{
		write = true;	 	
		t.write_metadata("tmp/table_" + std::to_string(tmp_table++),t.row_count,t1.columnNames.size()+t2.columnNames.size());
	}
	for(auto col_name1:t1.columnNames)
	{
		std::string col_name2 = col_name1;
		if(t2.columnNames.find(col_name1) != t2.columnNames.end())
			col_name2 = t1.get_column(col_name1).tname + "." + col_name1;
		column &col = t.new_column(col_name2);
		column &col1 = t1.get_column(col_name1);
		col.tname = col1.tname;
		if(col1.type)
		{
			col.type = 1;
			col.f.resize(t1.row_count * t2.row_count);
			assert(col1.f.size() == t1.row_count);

			typedef thrust::device_vector<float>::iterator Iterator;
			repeated_range<Iterator> Itr(col1.f.begin(), col1.f.end(), t2.row_count);
			thrust::copy(Itr.begin(), Itr.end(),col.f.begin());
		}else
		{
			col.type = 0;
			col.i.resize(t1.row_count * t2.row_count);
			assert(col1.i.size() == t1.row_count);
			
			typedef thrust::device_vector<int>::iterator Iterator;
			repeated_range<Iterator> Itr(col1.i.begin(), col1.i.end(), t2.row_count);
			thrust::copy(Itr.begin(), Itr.end(),col.i.begin());
		}
		if(write)
			t.write_column(col_name2);
	}
	
	for(auto col_name1 : t2.columnNames)
	{
		std::string col_name2 = col_name1;
		if(t1.columnNames.find(col_name1) != t1.columnNames.end())
			col_name2 = t2.get_column(col_name1).tname + "." + col_name1;
		column &col = t.new_column(col_name2);
		column &col2 = t2.get_column(col_name1);
		col.tname = col2.tname;
		if(col2.type)
		{
			col.type = 1;
			col.f.resize(t1.row_count * t2.row_count);
			assert(col2.f.size() == t2.row_count);
			
			typedef thrust::device_vector<float>::iterator Iterator;
			tiled_range<Iterator> Itr(col2.f.begin(), col2.f.end(), t1.row_count);
			thrust::copy(Itr.begin(), Itr.end(),col.f.begin());
		}else
		{
			col.type = 0;
			col.i.resize(t1.row_count * t2.row_count);
			assert(col2.i.size() == t2.row_count);
			
			typedef thrust::device_vector<int>::iterator Iterator;
			tiled_range<Iterator> Itr(col2.i.begin(), col2.i.end(), t1.row_count);
			thrust::copy(Itr.begin(), Itr.end(),col.i.begin());
		}
		if(write)
			t.write_column(col_name2);
	}


	return t;
}	

void eval_join(table &t1,table &t2,table &t3,table &t4,table &t5)
{
	//t1 corss product, t2 join condition, t3 first table, t4 second table, t5 join type
	if(t5.size() != 1 || t5.get_first_column().type != 0 || t5.get_first_column().i.size() != 1)
		yyerror("Invalid Join : error with join type");
	
	int type = t5.get_first_column().i[0];
	column &cond = t2.get_first_column();
	if(type == 1 || type == 2)
	{
		// cross product and inner join
		apply_result(t1,t2);
	}else if(type == 3)
	{
		// left outer join
		int N = t3.row_count;
		int K = t4.row_count;
		thrust::device_vector<int> sums(N,0);

		if(cond.type)
		{
			thrust::device_vector<float> &data = cond.f;
			assert(data.size() == N*K);
			thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
			thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(K)), data.begin(), thrust::discard_iterator<int>(), sums.begin());
			
			typedef thrust::device_vector<float>::iterator Iterator;
			strided_range<Iterator> it(data.begin(),data.end(),K);
			thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? 1 : a;});
		} else 
		{
			thrust::device_vector<int> &data = cond.i;
			assert(data.size() == N*K);
			thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
			thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(K)), data.begin(), thrust::discard_iterator<int>(), sums.begin());
			//thrust::transform(sums.begin(),sums.end(),sums.begin(),thrust::placeholders::_1 == 0);
			
			typedef thrust::device_vector<int>::iterator Iterator;
			strided_range<Iterator> it(data.begin(),data.end(),K);
			thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? 1 : a;});
		}
		for(auto cname:t4.columnNames)
		{
			if(t1.columnNames.find(cname) == t1.columnNames.end())
	 			cname = t4.get_column(cname).tname + "." + cname;
			column &col = t1.get_column(cname);
			if(col.type)
			{
				thrust::device_vector<float> &data = col.f;
				typedef thrust::device_vector<float>::iterator Iterator;
				strided_range<Iterator> it(data.begin(),data.end(),K);
				thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? FLOAT_FLAG : a;});
			}else
			{
				thrust::device_vector<int> &data = col.i;
				typedef thrust::device_vector<int>::iterator Iterator;
				strided_range<Iterator> it(data.begin(),data.end(),K);
				thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? INT_FLAG : a;});
			}
		}
		
		apply_result(t1,t2);
		
	}else if(type == 4)
	{
		//right outer join
		int N = t3.row_count;
		int K = t4.row_count;
		thrust::device_vector<int> sums(K,0);

		if(cond.type)
		{
			thrust::device_vector<float> &data = cond.f;
			assert(data.size() == N*K);
			thrust::device_vector<float> output(data.size());
			thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
			thrust::gather(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor2(N,K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor2(N,K)), data.begin(), output.begin());
			thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(N)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(N)), output.begin(), thrust::discard_iterator<int>(), sums.begin());
			
			thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? 1 : a;});
		} else 
		{
			thrust::device_vector<int> &data = cond.i;
			assert(data.size() == N*K);
			thrust::device_vector<int> output(data.size());
			thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
			thrust::gather(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor2(N,K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor2(N,K)), data.begin(), output.begin());
			thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(N)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(N)), output.begin(), thrust::discard_iterator<int>(), sums.begin());
			
			thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? 1 : a;});
		}
		for(auto cname:t3.columnNames)
		{
			if(t1.columnNames.find(cname) == t1.columnNames.end())
	 			cname = t3.get_column(cname).tname + "." + cname;
			column &col = t1.get_column(cname);
			if(col.type)
			{
				thrust::device_vector<float> &data = col.f;
				thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? FLOAT_FLAG : a;});
			}else
			{
				thrust::device_vector<int> &data = col.i;
				thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? INT_FLAG : a;});
			}
		}
		
		apply_result(t1,t2);
		
	}else if(type == 5)
	{
		//full outer join
		typedef struct dtypes
		{
			int i;
			float f;
			std::string s;
		}dtypes;
		std::unordered_map<std::string,dtypes> store_overlap;
		{
			// left outer join
			int N = t3.row_count;
			int K = t4.row_count;
			thrust::device_vector<int> sums(N,0);

			if(cond.type)
			{
				thrust::device_vector<float> &data = cond.f;
				assert(data.size() == N*K);
				thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
				thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(K)), data.begin(), thrust::discard_iterator<int>(), sums.begin());
				//thrust::transform(sums.begin(),sums.end(),sums.begin(),thrust::placeholders::_1 == 0);
				
				typedef thrust::device_vector<float>::iterator Iterator;
				strided_range<Iterator> it(data.begin(),data.end(),K);
				thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? 1 : a;});
				if(sums[0]==0)
					data[0] = 0;
			} else 
			{
				thrust::device_vector<int> &data = cond.i;
				assert(data.size() == N*K);
				thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
				thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(K)), data.begin(), thrust::discard_iterator<int>(), sums.begin());
				//thrust::transform(sums.begin(),sums.end(),sums.begin(),thrust::placeholders::_1 == 0);
				
				typedef thrust::device_vector<int>::iterator Iterator;
				strided_range<Iterator> it(data.begin(),data.end(),K);
				thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? 1 : a;});
				if(sums[0]==0)
					data[0] = 0;
			}
			
			if(sums[0]==0)
	 		{
				for(auto cname:t1.columnNames)
				{
					column &col = t1.get_column(cname);
					if(col.type)
						store_overlap[cname].f = col.f[0];
					else	
						store_overlap[cname].i = col.i[0];
				}
			}

			for(auto cname:t4.columnNames)
			{
				if(t1.columnNames.find(cname) == t1.columnNames.end())
					cname = t4.get_column(cname).tname + "." + cname;
				column &col = t1.get_column(cname);
				if(col.type)
				{
					thrust::device_vector<float> &data = col.f;
					typedef thrust::device_vector<float>::iterator Iterator;
					strided_range<Iterator> it(data.begin(),data.end(),K);
					thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? FLOAT_FLAG : a;});
					if(sums[0]==0)
		 				col.f[0] = store_overlap[cname].f, store_overlap[cname].f = FLOAT_FLAG;
				}else
				{
					thrust::device_vector<int> &data = col.i;
					typedef thrust::device_vector<int>::iterator Iterator;
					strided_range<Iterator> it(data.begin(),data.end(),K);
					thrust::transform(it.begin(),it.end(),sums.begin(),it.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? INT_FLAG : a;});
					if(sums[0]==0)
		 				col.i[0] = store_overlap[cname].i, store_overlap[cname].i = INT_FLAG;
				}
			}
			
		
		}
		
		{
			//right outer join
			int N = t3.row_count;
			int K = t4.row_count;
			thrust::device_vector<int> sums(K,0);

			if(cond.type)
			{
				thrust::device_vector<float> &data = cond.f;
				assert(data.size() == N*K);
				thrust::device_vector<float> output(data.size());
				thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
				thrust::gather(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor2(N,K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor2(N,K)), data.begin(), output.begin());
				thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(N)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(N)), output.begin(), thrust::discard_iterator<int>(), sums.begin());
				
				thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? 1 : a;});
			} else 
			{
				thrust::device_vector<int> &data = cond.i;
				assert(data.size() == N*K);
				thrust::device_vector<int> output(data.size());
				thrust::transform(data.begin(),data.end(),data.begin(),thrust::placeholders::_1 != 0);
				thrust::gather(thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor2(N,K)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor2(N,K)), data.begin(), output.begin());
				thrust::reduce_by_key(thrust::device, thrust::make_transform_iterator(thrust::counting_iterator<int>(0), functor1(N)), thrust::make_transform_iterator(thrust::counting_iterator<int>(N*K), functor1(N)), output.begin(), thrust::discard_iterator<int>(), sums.begin());
				
				thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? 1 : a;});
			}
			for(auto cname:t3.columnNames)
			{
				if(t1.columnNames.find(cname) == t1.columnNames.end())
					cname = t3.get_column(cname).tname + "." + cname;
				column &col = t1.get_column(cname);
				if(col.type)
				{
					thrust::device_vector<float> &data = col.f;
					thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (float &a,int &b){return (b == 0) ? FLOAT_FLAG : a;});
				}else
				{
					thrust::device_vector<int> &data = col.i;
					thrust::transform(data.begin(),data.begin() + K,sums.begin(),data.begin(),[=] __device__ __host__ (int &a,int &b){return (b == 0) ? INT_FLAG : a;});
				}
			}
			
			
		}
		if(store_overlap.size())
		{
			for(auto &p:store_overlap)
			{
				column &col = t1.get_column(p.first);
				if(col.type)
					col.f.push_back(p.second.f);
				else
					col.i.push_back(p.second.i);
			}
			t1.row_count += 1;
			t2.row_count += 1;
	 		if(cond.type)
				cond.f.push_back(1);
			else
				cond.i.push_back(1);
		}
		apply_result(t1,t2);

	}
	
}

void aggregate_function(table &t1, std::string agfunc, std::string new_name)
{
	if(t1.size() != 1)
		yyerror("aggregate function called on a table not column");
	column &opcol = t1.get_first_column();
	column &newcol = *(new column);
	if(opcol.type)
	{
	 	if( agfunc == "SUM")
		{
			float init = 0.0; 
			float sum = thrust::reduce(opcol.f.begin(), opcol.f.end(), init, thrust::plus<float>());
			opcol.f.clear();
			newcol.type = 1;
			newcol.f.push_back(sum);
		}
		else if(agfunc == "AVG")
		{
			float init = 0.0; 
			float sum = thrust::reduce(opcol.f.begin(), opcol.f.end(), init, thrust::plus<float>());
			float fsize = (float)opcol.f.size();
			opcol.f.clear();
			newcol.type = 1;
			newcol.f.push_back(sum/fsize);
		 }
		else if(agfunc == "COUNT")
		{
			int fcount = opcol.f.size(); 
			opcol.f.clear();
			newcol.type= 0;
			newcol.i.push_back(fcount);
		}else if(agfunc == "MAX")
		{
			float max = *thrust::max_element(opcol.f.begin(),opcol.f.end()); 
			opcol.f.clear();
			newcol.type= 1;
			newcol.f.push_back(max);
		}else if(agfunc == "MIN")
		{
			thrust::sort(opcol.f.begin(),opcol.f.end());
			thrust::device_vector<float>::iterator it;
			it = thrust::upper_bound(opcol.f.begin(),opcol.f.end(),FLOAT_FLAG);
			float min = FLOAT_FLAG;
			if(it != opcol.f.end())
				min = *it;
			opcol.f.clear();
			newcol.type= 1;
			newcol.f.push_back(min);
		}
		else
			yyerror(agfunc + " : No such aggregate function");
	}
	else
	{
	 	if( agfunc == "SUM")
		{
			int init = 0; 
			int sum = thrust::reduce(opcol.i.begin(), opcol.i.end(), init, thrust::plus<int>());
			opcol.i.clear();
			newcol.type = 0;
			newcol.i.push_back(sum);
		}
		else if(agfunc == "AVG")
		{
			int init = 0; 
			int sum = thrust::reduce(opcol.i.begin(), opcol.i.end(), init, thrust::plus<int>());
			float isize = (float)opcol.i.size();
			opcol.i.clear();
			newcol.type = 1;
			newcol.f.push_back(sum/isize);
		}
		else if(agfunc == "COUNT")
		{
			int icount = opcol.i.size(); 
			opcol.i.clear();
			newcol.type = 0;
			newcol.i.push_back(icount);
		}else if(agfunc == "MAX")
		{
			int max = *thrust::max_element(opcol.i.begin(),opcol.i.end()); 
			opcol.i.clear();
			newcol.type= 0;
			newcol.i.push_back(max);
		}else if(agfunc == "MIN")
		{
			thrust::sort(opcol.i.begin(),opcol.i.end());
			thrust::device_vector<int>::iterator it;
			it = thrust::upper_bound(opcol.i.begin(),opcol.i.end(),INT_FLAG);
			int min = INT_FLAG;
			if(it != opcol.i.end())
				min = *it;
			opcol.i.clear();
			newcol.type= 0;
			newcol.i.push_back(min);
		}
		else
			yyerror(agfunc + " : No such aggregate function");
	}
	
	t1.erase_column(t1.get_first_column_name());
	t1.row_count = 1;
	t1.copy_column(new_name,newcol);
}

void make_sorted(table &t2, std::string col_order, bool col_present, bool is_desc)
{
	column &order_on1 = t2.get_column(col_order);
	thrust::device_vector<int> new_order(t2.row_count);
	thrust::sequence(new_order.begin(),new_order.end(),0);
	if(order_on1.type)
	{
		if(is_desc)
			thrust::sort_by_key(order_on1.f.begin(),order_on1.f.end(),new_order.begin(),thrust::greater<float>());
		else
			thrust::sort_by_key(order_on1.f.begin(),order_on1.f.end(),new_order.begin());
	}
	else
	{
		if(is_desc)
			thrust::sort_by_key(order_on1.i.begin(),order_on1.i.end(),new_order.begin(),thrust::greater<int>());
		else
			thrust::sort_by_key(order_on1.i.begin(),order_on1.i.end(),new_order.begin());
	}
	
	auto iter_col = t2.columnNames.begin();
	thrust::device_vector<int> temp_i(t2.row_count);
	thrust::device_vector<float> temp_f(t2.row_count);

	while(iter_col != t2.columnNames.end())
	{
		if(*iter_col != col_order)
		{
			column &temp_col = t2.get_column(*iter_col);
			if(temp_col.type)
			{
				thrust::gather(new_order.begin(), new_order.end(), temp_col.f.begin(), temp_f.begin());
				temp_col.f = temp_f;
			}
			else
			{
				thrust::gather(new_order.begin(), new_order.end(), temp_col.i.begin(), temp_i.begin());
				temp_col.i = temp_i;
			}
		}
		iter_col++;
	}
	if(!col_present)
		t2.erase_column(col_order);
}
/*
void validate_columns(table &t3)
{
	assert(t3.size() == t3.columnNames.size());
	for(auto c:t3.columnNames)
	{
		column &col = t3.get_column(c);
		if(col.type)
		{
			if(col.f.size() != t3.row_count)
				if(col.f.size() == 1)
					col.f.resize(t3.row_count,col.f[0]);
				else
					yyerror("Invalid Column elements.");
		}else
		{
			if(col.i.size() != t3.row_count)
				if(col.i.size() == 1)
					col.i.resize(t3.row_count,col.i[0]);
				else
					yyerror("Invalid Column elements.");
		}
	}
}

bool check_keys(table &t)
{
	if(t.size() != 1)
		yyerror("Invalid keys.");
	column &col = t.get_first_column();
	int count = t.row_count;
	if(col.type)
		count -= thrust::count_if(col.f.begin(),col.f.end(),thrust::logical_not<float>());
	else	
		count -= thrust::count_if(col.i.begin(),col.i.end(),thrust::logical_not<int>());
	return (count != 0);
}
*/
table &eval(node *root,table &t)
{
	std::string &name = *(root->name);
	std::string &id = *(root->id);
	if(id == "cmd")
	{
		if(root->size == 1)
		{
			std::cout<<std::endl<<"Logging Out.\n"<<std::endl;
			exit(0);
		}else if(root->size == 2)
		{
			dbpath = *root->child[1]->name;
			if(dbpath == "")
				dbpath = "./";
			std::string cmd = "mkdir -p " + dbpath + "tmp/";
			int i = system(cmd.c_str());
			if(i != 0)
	 		{
				std::cout<<"Unable to create space for temporary table."<<std::endl;
				exit(0);
			}
			cmd = "rm -rf " + dbpath + "tmp/*";
			i = system(cmd.c_str());
			if(i != 0)
	 		{
				std::cout<<"Unable to clear the space for temporary table."<<std::endl;
				exit(0);
			}
			yyerror("Database changed");
		}
		table &t1 = eval(root->child[3],t);
		if(t1.row_count == 0)
			return t1;
		if(root->size > 6)
		{
			table &t2 = eval(root->child[5],t1);
			t1.updatekey(t2);
			if(t1.row_count == 0)
				return t1;
			
			table &t3 = eval(root->child[1],t1);
			
			std::string col_order = t3.get_column_name(*(root->child[7]->child[0]->name));
			bool col_present = true;
			if(t3.columnNames.find(col_order) == t3.columnNames.end())
			{
				table &t4 = eval(root->child[7]->child[0],t1);
				column &order_on = t4.get_first_column();
				t3.copy_column(col_order, order_on);
				col_present = false;
			}
			
			bool is_desc = false;
			if(root->child[7]->child[1])
			{				if(*(root->child[7]->child[1]->name) == "DESC")

		 			is_desc = true;		
			}
			make_sorted(t3,col_order,col_present,is_desc);
			return t3;	
		}
		else if(root->size == 6)
		{
			if(*(root->child[4]->id) == "ORDER_BY")
			{
				table &t2 = eval(root->child[1],t1);
				//validate_columns(t2);
				std::string col_order = t2.get_column_name(*(root->child[5]->child[0]->name));
				bool col_present = true;
				if(t2.columnNames.find(col_order) == t2.columnNames.end())
				{
					table &t3 = eval(root->child[5]->child[0],t1);
					column &order_on = t3.get_first_column();
					t2.copy_column(col_order, order_on);
					col_present = false;
				}
				bool is_desc = false;
				if(root->child[5]->child[1])
				{
					if(*(root->child[5]->child[1]->name) == "DESC")
		 				is_desc = true;		
				}
				make_sorted(t2,col_order,col_present,is_desc);
				return t2;	 	
			}
			else
			{
				table &t2 = eval(root->child[5],t1);
				t1.updatekey(t2);
				if(t1.row_count == 0)
					return t1;
				table &t3 = eval(root->child[1],t1);
				//validate_columns(t3);
				
				//use t2 as key to select rows from table t3

				//apply_result(t3,t2);
				return t3;
			}
		}else
		{
			table &t2 = eval(root->child[1],t1);
			return t2;
		}
	}else if(id == "columns")
	{
		if(root->size == 1)
		{
			table &t1 = eval(root->child[0],t);
			return t1;
		}
		table &t1 = eval(root->child[0],t);
		table &t2 = eval(root->child[2],t);
		if(t1.row_count != t2.row_count)
			yyerror("Aggregated column with nonaggregated colmns. (different number of rows)");
		t1.copy_column(*(root->child[2]->name),t2.get_column(*(root->child[2]->name)));
		
		return t1;
	}else if(id == "column")
	{
		if(*(root->child[0]->name) == "*")
		{
			for(auto col:t.columnNames)
				t.get_column(col);
			return t;
		}
		table &t1 = eval(root->child[0],t);
		if(root->size > 1)
			t1.move_column(*(root->child[0]->name),*(root->child[2]->name));
		return t1;
	}else if(id == "expr")
	{
		if(root->size == 3)
		{
			table &t1 = eval(root->child[0],t);
			table &t2 = eval(root->child[2],t);

			binary_op(t1,t2,*(root->child[1]->id));
			t1.move_column(t1.get_first_column_name(),name);
			return t1;	
		}else if(root->size == 2)
		{
			table &t1 = eval(root->child[1],t);
			unary_op(t1,*(root->child[1]->id));
			t1.move_column(t1.get_first_column_name(),name);
			return t1;
		}else
		{	
			return eval(root->child[0],t);
		}

	}else if(id == "Pexpr")
	{
		if(*(root->child[0]->id) == "aggregate")
		{	
			std::string agfunc = *(root->child[0]->name);
			std::string fname = *(root->name);
			if(root->size == 6)
			{
				if(t.name == *(root->child[2]->name))
				{
					table &t1	= *(new table);
					t1.name = t.name;
					t1.set_column(*(root->child[4]->name),t.get_column(*(root->child[4]->name)));
					aggregate_function(t1,agfunc,fname);
					return t1;
				}else 
				{
					if(t.columnNames.find(name) == t.columnNames.end())
					{
						yyerror(name + ": No such column in " + t.name);
					}else
					{
						table &t1 = *(new table);
						t1.name = t.name;
						t1.set_column(name,t.get_column(name));
						aggregate_function(t1,agfunc,fname);
						return t1;
					}
				}
			}else
			{
				if(*(root->child[2]->id) == "cmd")
				{
					table &t1 = eval(root->child[2],t);
					if(t1.size() == 1)
					{
						std::string cname = t1.get_first_column_name();
						t1.move_column(cname,*root->name);
						aggregate_function(t1,agfunc,fname);
					}
					else
						yyerror("subquery has more than one column");
					return t1;
				}else if(*(root->child[2]->id) == "expr")
				{
					table &t1 = eval(root->child[2],t);
					if(t1.size() == 1)
					{
						std::string cname = t1.get_first_column_name();
						t1.move_column(cname,*root->name);
						aggregate_function(t1,agfunc,fname);
					}
					else
						yyerror("subquery has more than one column");
					return t1;
				}
				else
				{
					if(t.columnNames.find(*(root->child[2]->name)) == t.columnNames.end())
					{
						yyerror("aggregate called on table");
					}else
					{
						table &t1 = *(new table);
						t1.name = t.name;
						t1.set_column(*(root->child[2]->name),t.get_column(*(root->child[2]->name)));
						aggregate_function(t1,agfunc,fname);
						return t1;
					}
				}
			}
		}
		else
		{	
			if(root->size == 3)
			{
				if(*(root->child[1]->id) == ".")
				{
					if(t.name == *(root->child[0]->name))
					{
						table &t1 = *(new table);
						t1.name = t.name;
						t1.set_column(*(root->child[2]->name),t.get_column(*(root->child[2]->name)));
						return t1;
					}else 
					{
						if(t.columnNames.find(name) == t.columnNames.end())
						{
							yyerror(name + ": No such column in " + t.name);
						}else
						{
							table &t1 = *(new table);
							t1.name = t.name;
							t1.set_column(name,t.get_column(name));
							return t1;
						}
					}
				}else
				{
					table &t1 = eval(root->child[1],t);
					t1.move_column(t1.get_first_column_name(),name);
					return t1;
				}
			}else
			{
				if(*(root->child[0]->id) == "integerLit")
				{
					table &t1 = *(new table);
					column col;
					col.type = 0;
					col.i.push_back(atoi(root->child[0]->name->c_str()));
					t1.set_column(*(root->child[0]->name),col);
					return t1;
				}else if(*(root->child[0]->id) == "floatLit")
				{
					table &t1 = *(new table);
					column col;
					col.type = 1;
					col.f.push_back(atof(root->child[0]->name->c_str()));
					t1.set_column(*(root->child[0]->name),col);
					return t1;
				}else
				{
					if(t.columnNames.find(*(root->child[0]->name)) == t.columnNames.end())
					{
						yyerror(*(root->child[0]->name) + " : No such column ");
					}else
					{
						table &t1 = *(new table);
						t1.name = t.name;
						t1.set_column(*(root->child[0]->name),t.get_column(*(root->child[0]->name)));
						return t1;
					}
				}
			}
		}
	}else if(id == "tables")
	{
		if(root->size == 5)
		{
			table &t1 = eval(root->child[0],t);
			table &t2 = eval(root->child[2],t1);
			
			table &t3 = eval(root->child[1],t);
			table &t4 = cross_prod(t1,t2);
			table &t5 = eval(root->child[4],t4);
			eval_join(t4,t5,t1,t2,t3);
			t4.name = name;
			return t4;
		}else if(root->size == 3)
		{
			table &t1 = eval(root->child[0],t);
			table &t2 = eval(root->child[2],t);
			
			table &t3 = eval(root->child[1],t);
			table &t4 = cross_prod(t1,t2);
			t4.name = name;
			return t4;
		}else
		{
			return eval(root->child[0],t);
		}
	}else if(id == "table")
	{
		if(root->size > 3)
		{
			table &t1 = eval(root->child[1],t);
			t1.name = *(root->child[4]->name);
			for(auto cname:t1.columnNames)
	 		{
				column &col = t1.get_column(cname);
				col.tname = t1.name;
			}
			return t1;
		}else if(root->size > 1)
		{
			table &t1 = *(new table(*(root->child[0]->name)));
			t1.name = *(root->child[2]->name);
			return t1;
		}else 
		{
			table &t1 = *(new table(name));
			return t1;
		}
	}else if(id == "join")
	{
		table &t1 = *(new table);
		if(*root->child[0]->id == ",")
		{
			column &col = t1.new_column(",");
			col.type = 0;
			col.i.push_back(1);
			t1.row_count = 1;
		}else if(*root->child[0]->id == "INNER JOIN")
		{
			column &col = t1.new_column("INNER JOIN");
			col.type = 0;
			col.i.push_back(2);
			t1.row_count = 1;
		}else if(*root->child[0]->id == "LEFT OUTER JOIN")
		{
			column &col = t1.new_column("LEFT OUTER JOIN");
			col.type = 0;
			col.i.push_back(3);
			t1.row_count = 1;
		}else if(*root->child[0]->id == "RIGHT OUTER JOIN")
		{
			column &col = t1.new_column("RIGHT OUTER JOIN");
			col.type = 0;
			col.i.push_back(4);
			t1.row_count = 1;
		}else if(*root->child[0]->id == "FULL OUTER JOIN")
		{
			column &col = t1.new_column("FULL OUTER JOIN");
			col.type = 0;
			col.i.push_back(5);
			t1.row_count = 1;
		}else
		{
			column &col = t1.new_column("ERROR TYPE");
			col.type = 0;
			col.i.push_back(0);
			t1.row_count = 1;
		}
		return t1;
	}
	return t;
}